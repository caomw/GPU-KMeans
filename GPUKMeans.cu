#include "hip/hip_runtime.h"
#include "GPUKMeans.h"
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/random.h>
#include <thrust/sort.h>
#include <thrust/functional.h>
#include <thrust/iterator/constant_iterator.h>
#include <iostream>

// ######################################################################
struct ClusterCompBinary :
  public thrust::binary_function<const ClassSummary&, const ClassSummary&, ClassSummary>
{
  size_t const clusterIdx;
  ClusterCompBinary(size_t const clusterIdx_) : clusterIdx(clusterIdx_) {}

  __host__ __device__
    ClassSummary operator()(const ClassSummary & p1, const ClassSummary & p2) const
    {
      ClassSummary result;

      if(p1.id == clusterIdx && p2.id == clusterIdx)
      {
        result.n = p1.n + p2.n;
        result.id = clusterIdx;
        result.mean_x = p1.mean_x + (p2.mean_x - p1.mean_x) * p2.n / result.n;
        result.mean_y = p1.mean_y + (p2.mean_y - p1.mean_y) * p2.n / result.n;
        result.mean_r = p1.mean_r + (p2.mean_r - p1.mean_r) * p2.n / result.n;
        result.mean_g = p1.mean_g + (p2.mean_g - p1.mean_g) * p2.n / result.n;
        result.mean_b = p1.mean_b + (p2.mean_b - p1.mean_b) * p2.n / result.n;
      }
      else if(p1.id == clusterIdx)
      {
        result.n = p1.n;
        result.id = p1.id;
        result.mean_x = p1.mean_x;
        result.mean_y = p1.mean_y;
        result.mean_r = p1.mean_r;
        result.mean_g = p1.mean_g;
        result.mean_b = p1.mean_b;
      }
      else if(p2.id == clusterIdx)
      {
        result.n = p2.n;
        result.id = p2.id;
        result.mean_x = p2.mean_x;
        result.mean_y = p2.mean_y;
        result.mean_r = p2.mean_r;
        result.mean_g = p2.mean_g;
        result.mean_b = p2.mean_b;
      }
      else
      {
        result.id = p1.id;
      }

      return result;
    }


};

// ######################################################################
struct ClusterCompUnary
{
  __device__
    ClassSummary operator()(const PointDescriptor& point) const
    {
      ClassSummary result;

      result.n = 1;
      result.id = point.classid;
      result.mean_x = point.x;
      result.mean_y = point.y;
      result.mean_r = point.r;
      result.mean_g = point.g;
      result.mean_b = point.b;

      return result;
    }
};

// ######################################################################
struct MinCluster
{
  __host__ __device__
    void operator()(thrust::tuple<PointDescriptor, float, ClassSummary> t)
    {
      PointDescriptor const & p = thrust::get<0>(t);

      float dist = sqrt(pow(thrust::get<2>(t).mean_x - p.x, 2) +
                        pow(thrust::get<2>(t).mean_y - p.y, 2) +
                        pow(thrust::get<2>(t).mean_r - p.r, 2) +
                        pow(thrust::get<2>(t).mean_g - p.g, 2) +
                        pow(thrust::get<2>(t).mean_b - p.b, 2));

      if(dist < thrust::get<1>(t))
      {
        thrust::get<1>(t) = dist;
        thrust::get<0>(t).classid = thrust::get<2>(t).id;
      }
    }
};

// ######################################################################
std::vector<ClassSummary> kmeans(std::vector<PointDescriptor> & points, size_t const k)
{
  // Randomly assign classes to each point descriptor
  thrust::default_random_engine rng;
  thrust::uniform_int_distribution<int> dist(0, k-1);
  std::vector<PointDescriptor>::iterator pointsit;
  for(pointsit = points.begin(); pointsit != points.end(); ++pointsit)
    pointsit->classid = dist(rng);

  // Create a device copy of the point descriptors
  thrust::device_vector<PointDescriptor> device_points(points.size());
  thrust::copy(points.begin(), points.end(), device_points.begin());

  // Create a device copy of the class means
  thrust::device_vector<ClassSummary> device_class_means(k);

  for(int iteration=0; iteration<10; ++iteration)
  {
    std::cout << "Iteration " << iteration << " --------------------------------" << std::endl;

    //Compute class means (cluster centers)
    for(size_t clusterIdx=0; clusterIdx<k; ++clusterIdx)
    {
      ClassSummary init;
      init.initialize();

      // Compute the mean for this cluster
      device_class_means[clusterIdx] =
        thrust::transform_reduce(device_points.begin(), device_points.end(), ClusterCompUnary(), init, ClusterCompBinary(clusterIdx));

      ClassSummary summary = device_class_means[clusterIdx];
      std::cout << "Cluster " << clusterIdx << " n: " << summary.n << 
        " x: " <<summary.mean_x << " y: " << summary.mean_y << " n: " << summary.n << 
        " r: " << summary.mean_r << " g: " << summary.mean_g << " b: " << summary.mean_b << std::endl;
    }

    // Assign each point to its new cluster
    thrust::device_vector<float>  distances(device_points.size(), std::numeric_limits<float>::max());
    for(size_t clusterIdx=0; clusterIdx<k; ++clusterIdx)
    {
      thrust::device_vector<float> clusterDistances(device_points.size());
      thrust::constant_iterator<ClassSummary> currentCluster(device_class_means[clusterIdx]);

      thrust::for_each(
          thrust::make_zip_iterator(thrust::make_tuple(device_points.begin(), distances.begin(), currentCluster)),
          thrust::make_zip_iterator(thrust::make_tuple(device_points.end(),   distances.end(),   currentCluster)),
          MinCluster());
    }
  }

  thrust::host_vector<ClassSummary> host_class_means(device_class_means.size());
  thrust::copy(device_class_means.begin(), device_class_means.end(), host_class_means.begin());
  std::vector<ClassSummary> class_means(host_class_means.size());
  std::copy(host_class_means.begin(), host_class_means.end(), class_means.begin());
  return class_means;
}

