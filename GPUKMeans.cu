#include "hip/hip_runtime.h"
#include "GPUKMeans.h"
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/random.h>
#include <thrust/sort.h>
#include <thrust/functional.h>
#include <iostream>

// ######################################################################
struct ClassSummary_unary_op
{
  __device__
    ClassSummary operator()(const pointdescriptor& point) const
    {
      ClassSummary result;

      result.n = 1;

      result.mean_x = point.x;
      result.mean_y = point.y;
      result.mean_r = point.r;
      result.mean_g = point.g;
      result.mean_b = point.b;

      return result;
    }
};

// ######################################################################
struct ClassSummary_binary_op : 
  public thrust::binary_function<const ClassSummary&, const ClassSummary&, ClassSummary>
{
  __host__ __device__
    ClassSummary operator()(const ClassSummary & p1, const ClassSummary & p2) const
    {
      ClassSummary result;

      size_t n = p1.n + p2.n;
      float const delta_x = p2.mean_x - p1.mean_x;
      float const delta_y = p2.mean_y - p1.mean_y;
      float const delta_r = p2.mean_r - p1.mean_r;
      float const delta_g = p2.mean_g - p1.mean_g;
      float const delta_b = p2.mean_b - p1.mean_b;

      result.n = n;
      result.mean_x = p1.mean_x + delta_x * p2.n / n;
      result.mean_y = p1.mean_y + delta_y * p2.n / n;
      result.mean_r = p1.mean_r + delta_r * p2.n / n;
      result.mean_g = p1.mean_g + delta_g * p2.n / n;
      result.mean_b = p1.mean_b + delta_b * p2.n / n;

      return result;
    }
};

// ######################################################################
struct ClassSorter :
  public thrust::binary_function<pointdescriptor const&, pointdescriptor const&, bool>
{
  __host__ __device__
    bool operator()(pointdescriptor const& p1, pointdescriptor const& p2)
    {
      return p1.classid < p2.classid;
    }
};

// ######################################################################
struct ClassFinder
{
  size_t classid;

  void setClass(size_t classid_)  { classid = classid_; }

  __host__ __device__
    bool operator()(pointdescriptor const& p)
    {
      return p.classid == classid;
    }
};

// ######################################################################
struct ClassAssigner
{
  ClassSummary * classesPtr;
  size_t const num_classes;
  float const float_max;

  __device__ __host__
  ClassAssigner(thrust::device_vector<ClassSummary> class_means_) :
    num_classes(class_means_.size()),
    float_max(1000000000000000)
  { 
  
    classesPtr = thrust::raw_pointer_cast(&class_means_[0]); 
  
  }

  __device__
    pointdescriptor operator()(pointdescriptor const& p) const
    {
      size_t min_idx = 0;
      float min_dist = float_max;
      for(size_t i=1; i<num_classes; ++i)
      {
        float dist = sqrt(pow(classesPtr[i].mean_x - p.x, 2) +
                          pow(classesPtr[i].mean_y - p.y, 2) +
                          pow(classesPtr[i].mean_r - p.r, 2) +
                          pow(classesPtr[i].mean_g - p.g, 2) +
                          pow(classesPtr[i].mean_b - p.b, 2));

        if(min_dist > dist)
        { min_dist = dist; min_idx = i; }
      }

      pointdescriptor ret = p;
      ret.classid = min_idx;
      return ret;
    }
};

// ######################################################################
std::vector<ClassSummary> kmeans(std::vector<pointdescriptor> & points, size_t k)
{
  // Randomly assign classes to each point descriptor
  thrust::default_random_engine rng;
  thrust::uniform_int_distribution<int> dist(0, k);
  std::vector<pointdescriptor>::iterator pointsit;
  for(pointsit = points.begin(); pointsit != points.end(); ++pointsit)
    pointsit->classid = dist(rng);

  // Create a device copy of the point descriptors
  thrust::device_vector<pointdescriptor> device_points(points.size());
  thrust::copy(points.begin(), points.end(), device_points.begin());

  thrust::device_vector<ClassSummary> device_class_means(k);

  for(int i=0; i<10; ++i)
  {
    // Sort the point descriptors by their class id
    ClassSorter sorter;
    thrust::sort(device_points.begin(), device_points.end(), sorter);

    // Find all of the class means
    thrust::device_vector<pointdescriptor>::iterator classBegin = device_points.begin();
    for(size_t classid = 0; classid < k; ++classid)
    {
      ClassFinder finder;
      finder.setClass(classid);
      thrust::device_vector<pointdescriptor>::iterator classEnd =
        thrust::find_if_not(classBegin, device_points.end(), finder);

      //Compute class means (cluster centers)
      ClassSummary init_summary;
      init_summary.initialize();
      ClassSummary_unary_op unary_summary_op;
      ClassSummary_binary_op binary_summary_op;

      device_class_means[classid] = thrust::transform_reduce(classBegin, classEnd,
          unary_summary_op, init_summary, binary_summary_op);
    }

    // Reassign all point descriptors to their nearest class
    ClassAssigner assigner(device_class_means);
    thrust::transform(device_points.begin(), device_points.end(), device_points.begin(), assigner);
  }

  thrust::host_vector<ClassSummary> host_class_means(device_class_means.size());
  thrust::copy(device_class_means.begin(), device_class_means.end(), host_class_means.begin());
  std::vector<ClassSummary> class_means(host_class_means.size());
  std::copy(host_class_means.begin(), host_class_means.end(), class_means.begin());
  return class_means;
}

